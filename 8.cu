#include "hip/hip_runtime.h"
﻿#include <array>
#include <cassert>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <>

static const std::filesystem::path INPUT_FILEPATH{ "img03.bmp" };
static constexpr auto HEADER_SIZE{ 54U };
static constexpr auto THREADS_PER_BLOCK{ 512U };

uint64_t proceed(const std::vector<char>& data);

int main()
{
	std::ifstream input(INPUT_FILEPATH, std::ios::in | std::ios::binary);
	if (!input.is_open())
	{
		std::cerr << "Failed to open file: path=" << INPUT_FILEPATH << '\n';
		return -1;
	}

	std::vector<char> data;
	{
		std::array<char, HEADER_SIZE> header;
		input.read(header.data(), header.size());

		const auto data_offset = *reinterpret_cast<uint32_t*>(&header[10]);
		const auto width = *reinterpret_cast<uint32_t*>(&header[18]);
		const auto height = *reinterpret_cast<uint32_t*>(&header[22]);
		std::cout << "data_offset=" << data_offset << ", width=" << width << ", height=" << height << '\n';

		data.resize(data_offset - HEADER_SIZE);
		input.read(data.data(), data.size());

		data.reserve(((width * 3 + 3) & (~3)) * height);

		std::vector<char> row_padded((width * 3 + 3) & (~3));
		for (size_t i{}; i < height; ++i)
		{
			input.read(row_padded.data(), row_padded.size());
			for (size_t j{}; j < width * 3; j += 3)
			{
				data.push_back(row_padded[j]);
				data.push_back(row_padded[j + 1]);
				data.push_back(row_padded[j + 2]);
			}
		}

		assert(data.size() % 3 == 0);
	}

	std::cout << "8b: cnt=" << proceed(data) << '\n';

	hipDeviceReset();

	return 0;
}

template <size_t BlockSize>
__global__ void cuda_proceed(char* data, const size_t data_size, uint64_t* cnt)
{
	const auto tid = threadIdx.x;
	const auto supply = (data_size % BlockSize) / 3;
	const auto spl_tmp = BlockSize - 1 - tid;
	const auto cull_data_size = data_size - supply * 3;
	const long end = data_size - 1 - (tid + 1) * cull_data_size / BlockSize - 3 * (spl_tmp < supply ? (supply - spl_tmp) : 0);
	const long start = data_size - 1 - tid * cull_data_size / BlockSize - 3 * (spl_tmp < supply ? (supply - spl_tmp - 1) : 0);
	uint64_t local_cnt{};
	for (long i = start; i > end; i -= 3)
		if (static_cast<size_t>(data[i]) * data[i - 1] * data[i - 2] < 1000)
			++local_cnt;

	__shared__ uint64_t shared_cnt[BlockSize];
	shared_cnt[tid] = local_cnt;
	__syncthreads();

	for (int reduction_size = BlockSize / 2; reduction_size; reduction_size >>= 1)
	{
		if (tid < reduction_size)
			shared_cnt[tid] += shared_cnt[tid + reduction_size];
		__syncthreads();
	}

	if (tid == 0)
		*cnt = shared_cnt[0];
}

uint64_t proceed(const std::vector<char>& data)
{
	char* dev_data;
	hipMalloc(&dev_data, data.size() * sizeof(char));
	hipMemcpy(dev_data, data.data(), data.size() * sizeof(char), hipMemcpyHostToDevice);

	uint64_t* dev_cnt;
	hipMalloc(&dev_cnt, sizeof(uint64_t));
	hipMemcpy(dev_data, data.data(), data.size() * sizeof(char), hipMemcpyHostToDevice);

	const dim3 block_size(THREADS_PER_BLOCK, 1, 1);
	const dim3 grid_size(1, 1, 1);
	cuda_proceed<THREADS_PER_BLOCK> << <grid_size, block_size >> > (dev_data, data.size(), dev_cnt);
	hipDeviceSynchronize();

	size_t cnt{};
	hipMemcpy(&cnt, dev_cnt, sizeof(uint64_t), hipMemcpyDeviceToHost);

	hipFree(dev_cnt);
	hipFree(dev_data);

	return cnt;
}